#include "hip/hip_runtime.h"
/**
 *  @file sat.cu
 *  @brief CUDA device code for GPU-Efficient Summed-Area Tables
 *  @author Andre Maximo
 *  @date September, 2011
 */

//== INCLUDES =================================================================

#include <symbol.h>

#include <dvector.h>

#include <gpufilter.h>
#include <gpuconsts.cuh>

#include <sat.cuh>

//== NAMESPACES ===============================================================

namespace gpufilter {

//== IMPLEMENTATION ===========================================================

//-- Algorithm SAT Stage 1 ----------------------------------------------------

__global__ __launch_bounds__( WS * SOW, MBO )
void algSAT_stage1( const float *g_in,
                    float *g_ybar,
                    float *g_vhat ) {

	const int tx = threadIdx.x, ty = threadIdx.y,
        bx = blockIdx.x, by = blockIdx.y, col = bx*WS+tx, row0 = by*WS;

	__shared__ float s_block[ WS ][ WS+1 ];

    float (*bdata)[WS+1] = (float (*)[WS+1]) &s_block[ty][tx];

	g_in += (row0+ty)*c_width+col;
	g_ybar += by*c_width+col;
	g_vhat += bx*c_height+row0+tx;

#pragma unroll
    for (int i = 0; i < WS-(WS%SOW); i+=SOW) {
        **bdata = *g_in;
        bdata += SOW;
        g_in += SOW * c_width;
    }
    if( ty < WS%SOW ) {
        **bdata = *g_in;
    }

	__syncthreads();

	if( ty == 0 ) {

        {   // calculate ybar -----------------------
            float (*bdata)[WS+1] = (float (*)[WS+1]) &s_block[0][tx];

            float prev = **bdata;
            ++bdata;

#pragma unroll
            for (int i = 1; i < WS; ++i, ++bdata)
                **bdata = prev = **bdata + prev;

            *g_ybar = prev;
        }

        {   // calculate vhat -----------------------
            float *bdata = s_block[tx];

            float prev = *bdata;
            ++bdata;

#pragma unroll
            for (int i = 1; i < WS; ++i, ++bdata)
                prev = *bdata + prev;

            *g_vhat = prev;
        }

	}

}

//-- Algorithm SAT Stage 2 ----------------------------------------------------

__global__ __launch_bounds__( WS * MW, MBO )
void algSAT_stage2( float *g_ybar,
                    float *g_ysum ) {

	const int tx = threadIdx.x, ty = threadIdx.y,
        bx = blockIdx.x, col0 = bx*MW+ty, col = col0*WS+tx;

	if( col >= c_width ) return;

	g_ybar += col;
	float y = *g_ybar;
	int ln = HWS+tx;

	if( tx == WS-1 )
		g_ysum += col0;

	volatile __shared__ float s_block[ MW ][ HWS+WS+1 ];

	if( tx < HWS ) s_block[ty][tx] = 0.f;
	else s_block[ty][ln] = 0.f;

	for (int n = 1; n < c_n_size; ++n) {

        // calculate ysum -----------------------

		s_block[ty][ln] = y;

		s_block[ty][ln] += s_block[ty][ln-1];
		s_block[ty][ln] += s_block[ty][ln-2];
		s_block[ty][ln] += s_block[ty][ln-4];
		s_block[ty][ln] += s_block[ty][ln-8];
		s_block[ty][ln] += s_block[ty][ln-16];

		if( tx == WS-1 ) {
			*g_ysum = s_block[ty][ln];
			g_ysum += c_m_size;
		}

        // fix ybar -> y -------------------------

		g_ybar += c_width;
		y = *g_ybar += y;

	}

}

//-- Algorithm SAT Stage 3 ----------------------------------------------------

__global__ __launch_bounds__( WS * MW, MBO )
void algSAT_stage3( const float *g_ysum,
                    float *g_vhat ) {

	const int tx = threadIdx.x, ty = threadIdx.y,
        by = blockIdx.y, row0 = by*MW+ty, row = row0*WS+tx;

	if( row >= c_height ) return;

	g_vhat += row;
	float y = 0.f, v = 0.f;

	if( row0 > 0 )
		g_ysum += (row0-1)*c_m_size;

	for (int m = 0; m < c_m_size; ++m) {

        // fix vhat -> v -------------------------

		if( row0 > 0 ) {
			y = *g_ysum;
			g_ysum += 1;
		}

		v = *g_vhat += v + y;
		g_vhat += c_height;

	}

}

//-- Algorithm SAT Stage 4 ----------------------------------------------------

__global__ __launch_bounds__( WS * SOW, MBO )
void algSAT_stage4( float *g_inout,
                    const float *g_y,
                    const float *g_v ) {

	const int tx = threadIdx.x, ty = threadIdx.y,
        bx = blockIdx.x, by = blockIdx.y, col = bx*WS+tx, row0 = by*WS;

	__shared__ float s_block[ WS ][ WS+1 ];

    float (*bdata)[WS+1] = (float (*)[WS+1]) &s_block[ty][tx];

	g_inout += (row0+ty)*c_width+col;
	if( by > 0 ) g_y += (by-1)*c_width+col;
	if( bx > 0 ) g_v += (bx-1)*c_height+row0+tx;

#pragma unroll
    for (int i = 0; i < WS-(WS%SOW); i+=SOW) {
        **bdata = *g_inout;
        bdata += SOW;
        g_inout += SOW * c_width;
    }
    if( ty < WS%SOW ) {
        **bdata = *g_inout;
    }

	__syncthreads();

	if( ty == 0 ) {

        {   // calculate y -----------------------
            float (*bdata)[WS+1] = (float (*)[WS+1]) &s_block[0][tx];

            float prev;
            if( by > 0 ) prev = *g_y;
            else prev = 0.f;

#pragma unroll
            for (int i = 0; i < WS; ++i, ++bdata)
                **bdata = prev = **bdata + prev;
        }

        {   // calculate x -----------------------
            float *bdata = s_block[tx];

            float prev;
            if( bx > 0 ) prev = *g_v;
            else prev = 0.f;

#pragma unroll
            for (int i = 0; i < WS; ++i, ++bdata)
                *bdata = prev = *bdata + prev;
        }

	}

	__syncthreads();

    bdata = (float (*)[WS+1]) &s_block[ty][tx];

	g_inout -= (WS-(WS%SOW))*c_width;

#pragma unroll
    for (int i = 0; i < WS-(WS%SOW); i+=SOW) {
        *g_inout = **bdata;
        bdata += SOW;
        g_inout += SOW * c_width;
    }
    if( ty < WS%SOW ) {
        *g_inout = **bdata;
    }

}

//-- Algorithm SAT Stage 4 (not-in-place) -------------------------------------

__global__ __launch_bounds__( WS * SOW, MBO )
void algSAT_stage4( float *g_out,
                    const float *g_in,
                    const float *g_y,
                    const float *g_v ) {

	const int tx = threadIdx.x, ty = threadIdx.y, bx = blockIdx.x, by = blockIdx.y, col = bx*WS+tx, row0 = by*WS;

	__shared__ float s_block[ WS ][ WS+1 ];

    float (*bdata)[WS+1] = (float (*)[WS+1]) &s_block[ty][tx];

	g_in += (row0+ty)*c_width+col;
	if( by > 0 ) g_y += (by-1)*c_width+col;
	if( bx > 0 ) g_v += (bx-1)*c_height+row0+tx;

#pragma unroll
    for (int i = 0; i < WS-(WS%SOW); i+=SOW) {
        **bdata = *g_in;
        bdata += SOW;
        g_in += SOW * c_width;
    }
    if( ty < WS%SOW ) {
        **bdata = *g_in;
    }

	__syncthreads();

	if( ty == 0 ) {

        {   // calculate y -----------------------
            float (*bdata)[WS+1] = (float (*)[WS+1]) &s_block[0][tx];

            float prev;
            if( by > 0 ) prev = *g_y;
            else prev = 0.f;

#pragma unroll
            for (int i = 0; i < WS; ++i, ++bdata)
                **bdata = prev = **bdata + prev;
        }

        {   // calculate x -----------------------
            float *bdata = s_block[tx];

            float prev;
            if( bx > 0 ) prev = *g_v;
            else prev = 0.f;

#pragma unroll
            for (int i = 0; i < WS; ++i, ++bdata)
                *bdata = prev = *bdata + prev;
        }

	}

	__syncthreads();

    bdata = (float (*)[WS+1]) &s_block[ty][tx];

	g_out += (row0+ty)*c_width+col;

#pragma unroll
    for (int i = 0; i < WS-(WS%SOW); i+=SOW) {
        *g_out = **bdata;
        bdata += SOW;
        g_out += SOW * c_width;
    }
    if( ty < WS%SOW ) {
        *g_out = **bdata;
    }

}

__global__ __launch_bounds__( WS * WS/4, MBO )
void algSAT_box( float *g_out,
                 float *g_sat,
                 float *g_in,
                 const int box_filter_radius)
{
	const int tx = threadIdx.x, ty = threadIdx.y, bx = blockIdx.x, by = blockIdx.y;

#pragma unroll
    for (int y=0; y<4; y++) {
        int col = bx*WS+tx;
        int row = by*WS+(4*ty+y);

        int z = box_filter_radius;
        int w = c_width-1;
        int h = c_height-1;

        int row_a = max(0,row-z-1), col_a = max(0,col-z-1);
        int row_b = min(h,row+z),   col_b = max(0,col-z-1);
        int row_c = max(0,row-z-1), col_c = min(w,col+z);
        int row_d = min(h,row+z),   col_d = min(w,col+z);

        float u = g_in [   row*c_width+col   ];
        float a = g_sat[ row_a*c_width+col_a ];
        float b = g_sat[ row_b*c_width+col_b ];
        float c = g_sat[ row_c*c_width+col_c ];
        float d = g_sat[ row_d*c_width+col_d ];

        if (row<z+1 || row>c_height-z-1 || col<z+1 || col>c_width-z-1) {
            g_out[row*c_width+col] = u;
        } else {
            g_out[row*c_width+col] = (a+d-c-b) / ((2*z+1)*(2*z+1));
        }
    }
}

//-- Host ---------------------------------------------------------------------

__host__
void prepare_algSAT( alg_setup& algs,
                     dvector<float>& d_inout,
                     dvector<float>& d_ybar,
                     dvector<float>& d_vhat,
                     dvector<float>& d_ysum,
                     const float *h_in,
                     const int& w,
                     const int& h ) {

    algs.width = w;
    algs.height = h;

    if( w % 32 > 0 ) algs.width += (32 - (w % 32));
    if( h % 32 > 0 ) algs.height += (32 - (h % 32));

    calc_alg_setup( algs, algs.width, algs.height );
    up_alg_setup( algs );

    d_inout.copy_from( h_in, w, h, algs.width, algs.height );

    d_ybar.resize( algs.n_size * algs.width );
    d_vhat.resize( algs.m_size * algs.height );
    d_ysum.resize( algs.m_size * algs.n_size );

}

__host__
void algSAT( dvector<float>& d_out,
             dvector<float>& d_ybar,
             dvector<float>& d_vhat,
             dvector<float>& d_ysum,
             const dvector<float>& d_in,
             const alg_setup& algs ) {

	const int nWm = (algs.width+MTS-1)/MTS, nHm = (algs.height+MTS-1)/MTS;
    const dim3 cg_img( algs.m_size, algs.n_size );
    const dim3 cg_ybar( nWm, 1 );
    const dim3 cg_vhat( 1, nHm );

    algSAT_stage1<<< cg_img, dim3(WS, SOW) >>>( d_in, d_ybar, d_vhat );

    algSAT_stage2<<< cg_ybar, dim3(WS, MW) >>>( d_ybar, d_ysum );

    algSAT_stage3<<< cg_vhat, dim3(WS, MW) >>>( d_ysum, d_vhat );

    algSAT_stage4<<< cg_img, dim3(WS, SOW) >>>( d_out, d_in, d_ybar, d_vhat );

}

__host__
void algSAT( dvector<float>& d_inout,
             dvector<float>& d_ybar,
             dvector<float>& d_vhat,
             dvector<float>& d_ysum,
             const alg_setup& algs ) {

	const int nWm = (algs.width+MTS-1)/MTS, nHm = (algs.height+MTS-1)/MTS;
    const dim3 cg_img( algs.m_size, algs.n_size );
    const dim3 cg_ybar( nWm, 1 );
    const dim3 cg_vhat( 1, nHm );

    algSAT_stage1<<< cg_img, dim3(WS, SOW) >>>( d_inout, d_ybar, d_vhat );

    algSAT_stage2<<< cg_ybar, dim3(WS, MW) >>>( d_ybar, d_ysum );

    algSAT_stage3<<< cg_vhat, dim3(WS, MW) >>>( d_ysum, d_vhat );

    algSAT_stage4<<< cg_img, dim3(WS, SOW) >>>( d_inout, d_ybar, d_vhat );

}

__host__
void algSAT( float *h_inout,
             const int& w,
             const int& h ) {

    alg_setup algs;
    dvector<float> d_out, d_ybar, d_vhat, d_ysum;

    prepare_algSAT( algs, d_out, d_ybar, d_vhat, d_ysum, h_inout, w, h );

    algSAT( d_out, d_ybar, d_vhat, d_ysum, algs );

    d_out.copy_to( h_inout, algs.width, algs.height, w, h );

}

__host__
void algBox( const int& box_filter_radius,
             dvector<float>& d_tmp,
             dvector<float>& d_box,
             dvector<float>& d_ybar,
             dvector<float>& d_vhat,
             dvector<float>& d_ysum,
             dvector<float>& d_in,
             const alg_setup& algs ) {

	const int nWm = (algs.width+MTS-1)/MTS, nHm = (algs.height+MTS-1)/MTS;
    const dim3 cg_img( algs.m_size, algs.n_size );
    const dim3 cg_ybar( nWm, 1 );
    const dim3 cg_vhat( 1, nHm );

    algSAT_stage1<<< cg_img, dim3(WS, SOW) >>>( d_in, d_ybar, d_vhat );

    algSAT_stage2<<< cg_ybar, dim3(WS, MW) >>>( d_ybar, d_ysum );

    algSAT_stage3<<< cg_vhat, dim3(WS, MW) >>>( d_ysum, d_vhat );

    algSAT_stage4<<< cg_img, dim3(WS, SOW) >>>( d_tmp, d_in, d_ybar, d_vhat );

    algSAT_box   <<< cg_img, dim3(WS, WS/4) >>>( d_box, d_tmp, d_in, box_filter_radius);

}
//=============================================================================
} // namespace gpufilter
//=============================================================================
