#include "hip/hip_runtime.h"
/**
 *  @file gpufilter.cu
 *  @brief CUDA device code for GPU-Efficient Recursive Filtering Algorithms
 *  @author Diego Nehab
 *  @author Andre Maximo
 *  @date September, 2011
 */

//== INCLUDES =================================================================

#include <cmath>
#include <cstdio>
#include <cfloat>
#include <cassert>
#include <iostream>
#include <algorithm>

#include <gputex.cuh>
#include <timer.h>

#include "sat.cu"
#include "alg4.cu"
#include "alg5.cu"

//== NAMESPACES ===============================================================

namespace gpufilter {

//== IMPLEMENTATION ===========================================================

//-- Host ---------------------------------------------------------------------

__host__
void gaussian_gpu( float **inout,
                   const int& w,
                   const int& h,
                   const int& d,
                   const float& s,
                   const int& extb,
                   const initcond& ic ) {
    float b10, a11, b20, a21, a22;
    weights1( s, b10, a11 );
    weights2( s, b20, a21, a22 );
    for (int c = 0; c < d; c++) {
        alg5( inout[c], w, h, b10, a11, extb, ic );
        alg4( inout[c], w, h, b20, a21, a22, extb, ic );
    }
}

__host__
void gaussian_gpu( float *inout,
                   const int& w,
                   const int& h,
                   const float& s,
                   const int& extb,
                   const initcond& ic ) {
    float b10, a11, b20, a21, a22;
    weights1( s, b10, a11 );
    weights2( s, b20, a21, a22 );
    alg5( inout, w, h, b10, a11, extb, ic );
    alg4( inout, w, h, b20, a21, a22, extb, ic );
}

__host__
void gaussian_gpu( float *inout,
                   const int& w,
                   const int& h,
                   const float& s,
                   float& runtime,
                   const int& extb,
                   const initcond& ic ) {
    float b10, a11, b20, a21, a22;
    weights1( s, b10, a11 );
    weights2( s, b20, a21, a22 );

    // alg5( inout, w, h, b10, a11, extb, ic );
    {

        alg_setup algs;
        dvector<float> d_out;
        dvector<float> d_transp_pybar, d_transp_ezhat, d_ptucheck, d_etvtilde;
        hipArray *a_in;

        prepare_alg5( algs, d_out, d_transp_pybar, d_transp_ezhat, d_ptucheck,
                d_etvtilde, a_in, inout, w, h, b10, a11, extb, ic );

        cpu_timer tm(0, "iP", true);
        alg5( d_out, d_transp_pybar, d_transp_ezhat, d_ptucheck, d_etvtilde,
                a_in, algs );
        hipDeviceSynchronize();
        tm.stop();
        runtime += tm.elapsed();

        d_out.copy_to( inout, w * h );

        hipFreeArray( a_in );
    }

    // alg4( inout, w, h, b20, a21, a22, extb, ic );
    {
        alg_setup algs, algs_transp;
        dvector<float> d_out, d_transp_out;
        dvector<float2> d_transp_pybar, d_transp_ezhat, d_pubar, d_evhat;
        hipArray *a_in;

        prepare_alg4( algs, algs_transp, d_out, d_transp_out, d_transp_pybar,
                d_transp_ezhat, d_pubar, d_evhat, a_in, inout, w, h,
                b20, a21, a22, extb, ic );

        cpu_timer tm(0, "iP", true);
        alg4( d_out, d_transp_out, d_transp_pybar, d_transp_ezhat, d_pubar,
                d_evhat, a_in, algs, algs_transp );
        hipDeviceSynchronize();
        tm.stop();
        runtime += tm.elapsed();

        d_out.copy_to( inout, w * h );

        hipFreeArray( a_in );
    }
}

__host__
void bspline3i_gpu( float **inout,
        const int& w,
        const int& h,
        const int& d,
        const int& extb,
        const initcond& ic ) {
    const float alpha = 2.f - sqrt(3.f);
    for (int c = 0; c < d; c++) {
        alg5( inout[c], w, h, 1.f+alpha, alpha, extb, ic );
    }
}

__host__
void bspline3i_gpu( float *inout,
        const int& w,
        const int& h,
        const int& extb,
        const initcond& ic ) {
    const float alpha = 2.f - sqrt(3.f);
    alg5( inout, w, h, 1.f+alpha, alpha, extb, ic );
}

//=============================================================================
} // namespace gpufilter
//=============================================================================
